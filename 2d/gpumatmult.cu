
#include <hip/hip_runtime.h>
const int threadsPerBlock = 128;

unsigned int hostOffsetSize;
unsigned int hostTargetSize;
unsigned int hostSourceSize;

static unsigned int is_set=0;
static unsigned int deviceOffsetSize;
static unsigned int deviceTargetSize;
static unsigned int deviceSourceSize;

static int   *deviceOffset;
static float *deviceTargetX;
static float *deviceTargetY;
static float *deviceTargetW;
static float *deviceSourceX;
static float *deviceSourceY;
static float *deviceSourceG;

__global__ void kernel(int* deviceOffset, float* deviceTargetX, float* deviceTargetY, float* deviceTargetW,
                       float sigma,       float* deviceSourceX, float* deviceSourceY, float* deviceSourceG)
{
  int i = blockIdx.x * threadsPerBlock + threadIdx.x;
  int jbase,jsize,jblok,j,jb,jj;
  float targetX,targetY,targetW,dx,dy,coef;
  __shared__ float sharedSourceX[threadsPerBlock];
  __shared__ float sharedSourceY[threadsPerBlock];
  __shared__ float sharedSourceG[threadsPerBlock];

  targetX = deviceTargetX[i];
  targetY = deviceTargetY[i];
  targetW = 0;
  coef = 0.5f/(sigma*sigma);
  jbase = deviceOffset[blockIdx.x];
  jsize = deviceOffset[blockIdx.x+1]-deviceOffset[blockIdx.x];
  jblok = (jsize + threadsPerBlock - 1) / threadsPerBlock;
  for (j = 0; j < jblok-1; j++) {
    jb = jbase + j * threadsPerBlock + threadIdx.x;
    __syncthreads();
    sharedSourceX[threadIdx.x] = deviceSourceX[jb];
    sharedSourceY[threadIdx.x] = deviceSourceY[jb];
    sharedSourceG[threadIdx.x] = deviceSourceG[jb];
    __syncthreads();
#pragma unroll 32
    for(jj = 0; jj < threadsPerBlock; jj++){
      dx = targetX-sharedSourceX[jj];
      dy = targetY-sharedSourceY[jj];
      targetW += sharedSourceG[jj]*exp(-(dx*dx+dy*dy)*coef);
    }
  }
  jb = jbase + j * threadsPerBlock + threadIdx.x;
  __syncthreads();
  sharedSourceX[threadIdx.x] = deviceSourceX[jb];
  sharedSourceY[threadIdx.x] = deviceSourceY[jb];
  sharedSourceG[threadIdx.x] = deviceSourceG[jb];
  __syncthreads();
  for(jj = 0; jj < jsize - (j * threadsPerBlock); jj++){
    dx = targetX-sharedSourceX[jj];
    dy = targetY-sharedSourceY[jj];
    targetW += sharedSourceG[jj]*exp(-(dx*dx+dy*dy)*coef);
  }
  deviceTargetW[i] = targetW/M_PI*coef;
}

void gpumatmult(float *hostTargetX, float *hostTargetY, float *hostTargetW,
                float *hostSourceX, float *hostSourceY, float *hostSourceG,
                int *hostOffset, int iblok, float sigma, int numCluster, int numTrunc)
{
  hostOffsetSize = sizeof(int) * (numCluster+1);
  hostTargetSize = sizeof(float) * numCluster * threadsPerBlock;
  hostSourceSize = sizeof(float) * numCluster * numTrunc;

  if (is_set==0) {
    hipSetDevice(0);
    is_set=1;
  }
  if (hostOffsetSize>deviceOffsetSize) {
    if(deviceOffsetSize!=0) hipFree(deviceOffset);
    hipMalloc((void**)&deviceOffset,hostOffsetSize);
    deviceOffsetSize=hostOffsetSize;
  }
  if (hostTargetSize>deviceTargetSize) {
    if(deviceTargetSize!=0) {
      hipFree(deviceTargetX);
      hipFree(deviceTargetY);
      hipFree(deviceTargetW);
    }
    hipMalloc((void**)&deviceTargetX,hostTargetSize);
    hipMalloc((void**)&deviceTargetY,hostTargetSize);
    hipMalloc((void**)&deviceTargetW,hostTargetSize);
    deviceTargetSize=hostTargetSize;
  }
  if (hostSourceSize>deviceSourceSize) {
    if(deviceSourceSize!=0) {
      hipFree(deviceSourceX);
      hipFree(deviceSourceY);
      hipFree(deviceSourceG);
    }
    hipMalloc((void**)&deviceSourceX,hostSourceSize);
    hipMalloc((void**)&deviceSourceY,hostSourceSize);
    hipMalloc((void**)&deviceSourceG,hostSourceSize);
    deviceSourceSize=hostSourceSize;
  }

  hipMemcpy(deviceOffset,hostOffset,hostOffsetSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceTargetX,hostTargetX,hostTargetSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceTargetY,hostTargetY,hostTargetSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceSourceX,hostSourceX,hostSourceSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceSourceY,hostSourceY,hostSourceSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceSourceG,hostSourceG,hostSourceSize,hipMemcpyHostToDevice);

  dim3 block(threadsPerBlock);
  dim3 grid(iblok);
  kernel<<< grid, block >>>(deviceOffset,deviceTargetX,deviceTargetY,deviceTargetW,
                                   sigma,deviceSourceX,deviceSourceY,deviceSourceG);

  hipMemcpy(hostTargetW,deviceTargetW,hostTargetSize,hipMemcpyDeviceToHost);

}
